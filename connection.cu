#include "hip/hip_runtime.h"

#include "main.hu"

using namespace std;

__host__
Connection::Connection (Neuron* neuron, int delay_init, float multiplier_init) {
    connected_neuron = neuron;
    // allocate
    hipMallocManaged(&memblock, (3+MAX_QUEUE)*sizeof(float*)); //stores everythting
    hipMallocManaged(&memblock_buffer, (2+MAX_QUEUE)*sizeof(float));
    multiplier = &(memblock_buffer[0]);
    delay = &(memblock_buffer[1]);
    state_queue = &(memblock_buffer[2]);
    // initialize
    connected_neuron_state = neuron->state;
    (*multiplier) = multiplier_init;
    (*delay) = float(delay_init);
    for (int i=0; i<(*delay); i++)
        state_queue[i] = 0.;
    // connect to memblock
    memblock[0] = connected_neuron_state;
    memblock[1] = multiplier;
    memblock[2] = delay;
    memblock[3] = state_queue;
    // synchronize
    hipDeviceSynchronize();
}
__host__
Connection::~Connection() {
    hipFree(memblock_buffer);
    hipFree(memblock);
    // synchronize
    hipDeviceSynchronize();
}

__device__
void Connection__time_step (float** connection_memblock) {
    // read from memblock
    float* connected_neuron_state = connection_memblock[0];
    float* multiplier = connection_memblock[1];
    float* delay = connection_memblock[2];
    float* state_queue = connection_memblock[3];
    // update the state queue
    for (int i=1; i<(*delay); i++) {
        state_queue[i-1] = state_queue[i];
    }
    state_queue[int(*delay)-1] = (*connected_neuron_state);
    // prepare head state (dendrite) for readout
    state_queue[0] *= (*multiplier);
}

__host__
ostream& operator<< (ostream& cout, const Connection& c) {
    hipDeviceSynchronize();
    cout << " --Conn. to " << *(c.connected_neuron->name) << ": multiplier=" << *(c.multiplier) << ", delay=" << *(c.delay) << ", queue: ";
    for (int i=0; i<*(c.delay); i++)
        cout << c.state_queue[i] << " ";
    cout << endl;
    return cout;
}

// utility kernels
__global__
void time_step_single_connection (float** connection_memblock) {
    Connection__time_step (connection_memblock);
}