
#include "main.hu"

using namespace std;

int main (void) {

    // initialize random number generator on host and device
    srand(time(NULL));
    hiprandState *curand_state;
    hipMalloc(&curand_state, sizeof(hiprandState));

    Brain bob ("bob", 1000, 100, 8, 4, 0., 10., 1, 10, 0.02);
    adjust_state_single_neuron<<<1,1>>>(bob.neurons[0]->state, 1.0);

    //cout << bob; return 0;

    for (int t=0; t<100; t++) {
        Brain__time_step_connections<<<40,32>>>(bob.n_connections, bob.connection_memblocks);
        hipDeviceSynchronize(); // wait until all connections updated
        Brain__time_step_neurons<<<40,32>>>(bob.n_neurons, bob.neuron_memblocks, bob.n_inputs);
        hipDeviceSynchronize(); // wait until all connections updated
        //cout << bob;
        bob.print_output();
        cout << "Time " << t << " finished." << endl << endl;
        Brain__shake_connections<<<40,32>>>(bob.n_connections, bob.connection_memblocks, 0.01, curand_state, time(NULL));
    }

    //cout << bob;

    /*Neuron n1 ("1"), n2 ("2");

    cout << n1 << n2;
    n2.attach_dendrite(&n1);
    Connection* c1 = n2.dendrites[0];
    adjust_state_single_neuron<<<1,1>>>(n1.state, 1.0);
    cout << n1 << n2;

    for (int time=0; time<10; time++) {
        time_step_single_connection<<<1,1>>>(c1->memblock);
        time_step_single_neuron<<<1,1>>>(n1.memblock);
        time_step_single_neuron<<<1,1>>>(n2.memblock);
        cout << n1 << n2;
    }*/

    // cleanup
    //hiprandDestroyGenerator();

    return 0;
}