#include "hip/hip_runtime.h"

#include "main.hu"

using namespace std;

__host__
Neuron::Neuron (string _name, float _state, float _decay_rate) {
    hipMallocManaged(&name, sizeof(string));
    (*name) = _name;
    // allocate
    dendrites = new Connection* [MAX_DENDRITES];
    hipMallocManaged(&memblock, (4+MAX_DENDRITES)*sizeof(float*)); //stores everythting
    hipMallocManaged(&memblock_buffer, 3*sizeof(float));
    state = &(memblock_buffer[0]);
    decay_rate = &(memblock_buffer[1]);
    n_dendrites = &(memblock_buffer[2]);
    // initialize
    (*state) = _state;
    (*decay_rate) = _decay_rate;
    (*n_dendrites) = 0;
    // connect to memblock
    memblock[0] = state;
    memblock[1] = decay_rate;
    memblock[2] = n_dendrites;
    dendrites_states = &(memblock[3]);
    // synchronize
    hipDeviceSynchronize();
}
__host__
Neuron::~Neuron () {
    for (int i=0; i<int(*n_dendrites); i++) {
        delete dendrites[i];
    }
    delete dendrites;
    hipFree(memblock_buffer);
    hipFree(memblock);
    hipFree(name);
    // synchronize
    hipDeviceSynchronize();
}

__device__
void Neuron__time_step (float** memblock) {
    float* state = memblock[0];
    float* decay_rate = memblock[1];
    float* n_dendrites = memblock[2];
    float** dendrites_states = &(memblock[3]);
    // decay previous state
    (*state) *= exp(-1./(*decay_rate));
    // read from the dendrites
    for (int i=0; i<int(*n_dendrites); i++) {
        Neuron__adjust_state(state, *(dendrites_states[i]));
    }
    Neuron__activate(state);
}

__device__
void Neuron__adjust_state (float* state, float dx) {
    (*state) += dx;
}
__device__
void Neuron__activate (float* state) {
    // use sigmoid activation
    (*state) = 2./(1.+exp(-(*state))) - 1.;
}

__device__
float Neuron__get_state (float* state) {
    return 1./(1.+exp(-(*state)));
}

__host__
void Neuron::attach_dendrite (Neuron* neuron, int delay_init, float multiplier_init) {
    hipDeviceSynchronize();
    dendrites[int(*n_dendrites)] = new Connection(neuron, max(1,delay_init), multiplier_init);
    dendrites_states[int(*n_dendrites)] = dendrites[int(*n_dendrites)]->state_queue;
    (*n_dendrites) += 1.0;
    hipDeviceSynchronize();
}

__host__
ostream& operator<< (ostream& cout, const Neuron& n) {
    hipDeviceSynchronize();
    cout << "Neuron" << *(n.name) << ": state=" << *(n.state) << ", decay_rate=" << *(n.decay_rate) << ", n_dendrites=" << *(n.n_dendrites) << endl;
    for (int i=0; i<int(*(n.n_dendrites)); i++) {
        cout << *(n.dendrites[i]);
    }
    return cout;
}

// utility kernels
__global__
void adjust_state_single_neuron (float* state, float x) {
    Neuron__adjust_state(state, x);
}
__global__
void time_step_single_neuron (float** neuron_memblock) {
    Neuron__time_step (neuron_memblock);
}