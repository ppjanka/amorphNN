#include "hip/hip_runtime.h"

#include "main.hu"

using namespace std;

__host__
Neuron::Neuron (string _name) {
    hipMallocManaged(&name, sizeof(string));
    (*name) = _name;
    hipMallocManaged(&state, sizeof(float));
    (*state) = 0.;
    hipMallocManaged(&decay_rate, sizeof(float));
    (*decay_rate) = 10.;
    hipMallocManaged(&n_dendrites, sizeof(int));
    (*n_dendrites) = 0;
    hipMallocManaged(&dendrites, MAX_DENDRITES * sizeof(Connection*));
    hipMallocManaged(&dendrites_states, MAX_DENDRITES * sizeof(float));
    hipDeviceSynchronize();
}
__host__
Neuron::~Neuron () {
    for (int i=0; i<*n_dendrites; i++) {
        delete dendrites[i];
    }
    hipFree(dendrites);
    hipFree(dendrites_states);
    hipFree(name);
    hipFree(state);
    hipFree(decay_rate);
    hipFree(n_dendrites);
}

__device__
void Neuron__time_step (float* state, float* decay_rate, int* n_dendrites, float** dendrites_states) {
    // decay previous state
    (*state) *= exp(-1./(*decay_rate));
    // read from the dendrites
    for (int i=0; i<(*n_dendrites); i++) {
        Neuron__adjust_state(state, *(dendrites_states[i]));
    }
}

__device__
void Neuron__adjust_state (float* state, float dx) {
    (*state) += dx;
}

__device__
float Neuron__get_state (float* state) {
    // use sigmoid activation
    return 1./(1.+exp(-(*state)));
}

__host__
void Neuron::attach_dendrite (Neuron* neuron) {
    hipDeviceSynchronize();
    dendrites[*n_dendrites] = new Connection(neuron);
    dendrites_states[*n_dendrites] = dendrites[*n_dendrites]->state_queue;
    (*n_dendrites)++;
    hipDeviceSynchronize();
}

__host__
ostream& operator<< (ostream& cout, const Neuron& n) {
    hipDeviceSynchronize();
    cout << "Neuron" << *(n.name) << ": state=" << *(n.state) << ", decay_rate=" << *(n.decay_rate) << ", n_dendrites=" << *(n.n_dendrites) << endl;
    for (int i=0; i<*(n.n_dendrites); i++) {
        cout << *(n.dendrites[i]);
    }
    return cout;
}