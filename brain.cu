#include "hip/hip_runtime.h"

#include "main.hu"

using namespace std;

__host__
Brain::Brain (string _name, int _n_neurons, int avg_connections_per_neuron, int _n_inputs, int _n_outputs, float max_init_state, float max_init_delay) {
    cout << "Creating " << _name << "'s brain..." << endl;
    // parse parameters
    cout << " -- parsing parameters.. " << flush;
    hipMallocManaged(&name, sizeof(string));
    (*name) = _name;
    hipMallocManaged(&n_neurons, sizeof(int));
    (*n_neurons) = _n_neurons;
    hipMallocManaged(&n_connections, sizeof(int));
    (*n_connections) = (_n_neurons-_n_inputs) * avg_connections_per_neuron;
    hipMallocManaged(&n_inputs, sizeof(int));
    (*n_inputs) = _n_inputs;
    hipMallocManaged(&n_outputs, sizeof(int));
    (*n_outputs) = _n_outputs;
    cout << "done." << endl;
    // allocate
    cout << " -- allocating memory.. " << flush;
    hipMallocManaged(&input_states, (*n_inputs) * sizeof(float*));
    hipMallocManaged(&output_states, (*n_outputs) * sizeof(float*));
    neurons = new Neuron* [*n_neurons];
    connections = new Connection* [*n_connections];
    hipMallocManaged(&neuron_memblocks, (*n_neurons) * sizeof(float**));
    hipMallocManaged(&connection_memblocks, (*n_connections) * sizeof(float**));
    cout << "done." << endl;
    // create neurons
    cout << " -- creating neurons.. " << flush;
    for (int i=0; i < (*n_neurons); i++) {
        // use random state and decay rate
        neurons[i] = new Neuron (to_string(i), max_init_state*rand()/RAND_MAX, max_init_delay*rand()/RAND_MAX);
        neuron_memblocks[i] = neurons[i]->memblock;
    }
    input_neurons = neurons;
    output_neurons = &(neurons[(*n_neurons)-(*n_outputs)-1]);
    for (int i=0; i < (*n_inputs); i++) {
        input_states[i] = neurons[i]->state;
    }
    int istart = (*n_neurons-*n_outputs);
    for (int i=istart; i < (*n_neurons); i++) {
        output_states[i-istart] = neurons[i]->state;
    }
    cout << "done." << endl;
    // wire the brain
    cout << " -- wiring the brain.. " << flush;
    int j = 0;
    int ci = 0;
    for (int i=(*n_inputs); i < (*n_neurons); i++) { // don't connect input neurons
        for (int k=0; k<avg_connections_per_neuron; k++) {// const conn per neur for now
            // randomly choose the connected neuron
            do {j = rand() % (*n_neurons);} while (j == i);
            neurons[i]->attach_dendrite(neurons[j]);
            connections[ci] = neurons[i]->dendrites[int(*(neurons[i]->n_dendrites))-1];
            connection_memblocks[ci] = connections[ci]->memblock;
            ci++;
        }
    }
    cout << "done." << endl;
    cout << " -- synchronizing.. " << flush;
    hipDeviceSynchronize();
    cout << "done." << endl;
    cout << _name << "'s brain is ready!" << endl << endl;
}

__host__
Brain::~Brain () {
    for (int i=0; i<(*n_neurons); i++) {
        delete neurons[i]; // also deletes connections
    }
    delete neurons;
    delete connections;
    hipFree(connection_memblocks);
    hipFree(neuron_memblocks);
    hipFree(input_states);
    hipFree(output_states);
    hipFree(n_connections);
    hipFree(n_neurons);
    hipFree(n_inputs);
    hipFree(n_outputs);
    hipFree(name);
    // synchronize
    hipDeviceSynchronize();
}

__global__
void Brain__time_step_connections (int* n_connections, float*** connection_memblocks) {
    // CUDA setup
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // update connections
    for (int i=index; i<(*n_connections); i+=stride) {
        Connection__time_step(connection_memblocks[i]);
    }
}
__global__
void Brain__time_step_neurons (int* n_neurons, float*** neuron_memblocks, int* n_inputs) {
    // CUDA setup
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // update neurons (ignore inputs)
    for (int i=index; i<(*n_neurons-*n_inputs); i+=stride) {
        Neuron__time_step(neuron_memblocks[i+(*n_inputs)]);
    }
}

// raise all connection multiplicators to a small random power in (1-eps, 1+eps)
__global__
void Brain__shake_connections (int* n_connections, float*** connection_memblocks, float eps, hiprandState* curand_state, unsigned long seed) {
    // CUDA setup
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // CUrand setup
    hiprand_init(seed, index, 0, &curand_state[index]);
    // update connections
    float power;
    for (int i=index; i<(*n_connections); i+=stride) {
        power = (1.-eps) + 2.*eps * hiprand_normal(curand_state);
        *(connection_memblocks[i][1]) = pow(*(connection_memblocks[i][1]), power);
    }
}
// raise all connection multiplicators to a given constant power
__global__
void Brain__feedback_connections (int* n_connections, float*** connection_memblocks, float power) {
    // CUDA setup
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // update connections
    for (int i=index; i<(*n_connections); i+=stride) {
        *(connection_memblocks[i][1]) = pow(*(connection_memblocks[i][1]), power);
    }
}

__host__
ostream& operator<< (ostream& cout, const Brain& b) {
    hipDeviceSynchronize();
    cout << "Brain " << *(b.name) << ":" << endl;
    for (int i=0; i<(*(b.n_neurons)); i++) {
        cout << "   " << *(b.neurons[i]);
    }
    cout << endl;
    return cout;
}
__host__
void Brain::print_output () {
    hipDeviceSynchronize();
    cout << *name << " says: " << flush;
    for (int i=0; i<(*n_outputs); i++) {
        cout << *(output_states[i]) << "  " << flush;
    }
    cout << endl;
}